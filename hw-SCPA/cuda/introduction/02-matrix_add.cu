#include "hip/hip_runtime.h"
// Copyright 2014, Cranfield University
// All rights reserved
// Author: Michał Czapiński (mczapinski@gmail.com)
//
// Adds two matrices on the GPU. Matrices are stored in linear memory in row-major order,
// i.e. A[i, j] is stored in i * COLS + j element of the vector.

#include <iostream>

#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>   // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers

#include <stdlib.h>

// Matrix dimensions. Can you make these input arguments? [DEFAULT: 4096]
int ROWS;
int COLS;

// TODO(later) Play a bit with the block size. Is 16x16 setup the fastest possible?
// Note: For meaningful time measurements you need sufficiently large matrix.
const dim3 BLOCK_DIM(16, 16);

// Simple CPU implementation of matrix addition.
void CpuMatrixAdd(int rows, int cols, const float* A, const float* B, float* C) {
  for (int row = 0; row < rows; ++row) {
    for (int col = 0; col < cols; ++col) {
      int idx = row * cols + col;
      C[idx] = A[idx] + B[idx];
    }
  }
}

// GPU implementation of matrix add using one CUDA thread per vector element.
__global__ void GpuMatrixAdd(int rows, int cols, const float* A, const float* B, float* C) {
  // FANFA: Calculate indices of matrix elements added by this thread. Assume 2D grid of blocks.
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  // TODO(later) Does it matter if you index rows with x or y dimension of threadIdx and blockIdx?

  // FANFA: Calculate the element index in the global memory and add the values.
  int idx = col + row * blockDim.x * gridDim.x;

  // FANFA: Make sure that no threads access memory outside the allocated area.
  if(idx < rows*cols) {
    C[idx] = A[idx] + B[idx];
  }

}

int main(int argc, char** argv) {

// ------------------- FANFA: Input arguments acquisition ------------------- //

  if(argc < 3) {
    std::cout << "Usage: ./matrix_add ROWS COLS" << std::endl;
    return -1;
  }

  ROWS = atoi(argv[1]);
  COLS = atoi(argv[2]);

  if(ROWS <= 0 || COLS <= 0) {
    std::cout << "Invalid input. Please try again." << std::endl;
    return -1;
  }

  // Variables useful for GRID_DIM definition.
  int grid_dim_1 = ROWS/16;
  int grid_dim_2 = COLS/16;

// ----------------------- Host memory initialisation ----------------------- //

  float* h_A = new float[ROWS * COLS];
  float* h_B = new float[ROWS * COLS];
  float* h_C = new float[ROWS * COLS];

  srand(time(0));
  for (int row = 0; row < ROWS; ++row) {
    for (int col = 0; col < COLS; ++col) {
      int idx = row * COLS + col;
      h_A[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
      h_B[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
    }
  }

// ---------------------- Device memory initialisation ---------------------- //

  // FANFA: Allocate global memory on the GPU.
  float *d_A, *d_B, *d_C;

  checkCudaErrors(hipMalloc((void **)&d_A, ROWS*COLS*sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_B, ROWS*COLS*sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_C, ROWS*COLS*sizeof(float)));

  // FANFA: Copy matrices from the host (CPU) to the device (GPU).
  checkCudaErrors(hipMemcpy(d_A, h_A, ROWS*COLS*sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_B, h_B, ROWS*COLS*sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_C, h_C, ROWS*COLS*sizeof(float), hipMemcpyHostToDevice));

// ------------------------ Calculations on the CPU ------------------------- //

  // Create the CUDA SDK timer.
  StopWatchInterface* timer = 0;
  sdkCreateTimer(&timer);

  timer->start();
  CpuMatrixAdd(ROWS, COLS, h_A, h_B, h_C);

  timer->stop();
  std::cout << "CPU time: " << timer->getTime() << " ms." << std::endl;

// ------------------------ Calculations on the GPU ------------------------- //

  // FANFA: Calculate the dimension of the grid of blocks (2D).
  const dim3 GRID_DIM(grid_dim_1, grid_dim_2);

  timer->reset();
  timer->start();
  GpuMatrixAdd<<<GRID_DIM, BLOCK_DIM>>>(ROWS, COLS, d_A, d_B, d_C);
  checkCudaErrors(hipDeviceSynchronize());

  timer->stop();
  std::cout << "GPU time: " << timer->getTime() << " ms." << std::endl;

  // FANFA: Download the resulting matrix d_C from the device and store it in h_A.
  checkCudaErrors(hipMemcpy(h_A, d_C, ROWS*COLS*sizeof(float), hipMemcpyDeviceToHost));

  // Now let's check if the results are the same.
  float diff = 0.0f;
  for (int row = 0; row < ROWS; ++row) {
    for (int col = 0; col < COLS; ++col) {
      int idx = row * COLS + col;
      diff = std::max(diff, std::abs(h_A[idx] - h_C[idx]));
    }
  }
  std::cout << "Max diff = " << diff << std::endl;  // Should be (very close to) zero.

// ------------------------------- Cleaning up ------------------------------ //

  delete timer;

  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));

  delete[] h_A;
  delete[] h_B;
  delete[] h_C;

  return 0;
}
